#include "hip/hip_runtime.h"
#include "../../include/0_CU/cu1_init.cuh"

//#define DATA_SIZE 1048576
#define THREAD_NUM 256
#define BLOCK_NUM 32

void printDeviceProp( const hipDeviceProp_t &prop )
{
    printf( "Device name: %s.\n", prop.name);
    printf( "totalGlobalMem: %d.\n", prop.totalGlobalMem);
    printf( "sharedMemPerBlock: %d.\n", prop.sharedMemPerBlock );
    printf( "regsPerBlock: %d.\n", prop.regsPerBlock );
    printf( "warpSize: %d.\n", prop.warpSize );
    printf( "memPitch: %d.\n", prop.memPitch );
    printf( "maxThreadsPerBlock: %d.\n", prop.maxThreadsPerBlock );
    printf( "maxThreadsDim[0 - 2]: %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
    printf( "maxGridSize[0 - 2]: %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] );
    printf( "totalConstMem: %d.\n", prop.totalConstMem );
    printf( "major.minor: %d, %d.\n", prop.major, prop.minor );
    printf( "clockRate: %d.\n", prop.clockRate );
    printf( "textureALignment: %d.\n", prop.textureAlignment );
    printf( "deviceOverlap: %d.\n", prop.deviceOverlap );
    printf( "multiProcessorCount: %d.\n", prop.multiProcessorCount );
}

bool InitCUDA()
{
    int count ;
    hipGetDeviceCount( &count );

    if( count == 0 ) return false;

    int i;
    for( i = 0; i < count; i++ )
    {
        hipDeviceProp_t prop;
        if( hipGetDeviceProperties(&prop, i) == hipSuccess )
        {
            printDeviceProp( prop );
            if( prop.major >= 1 ) break;
        }
    }

    if( i == count ) return false;

    hipSetDevice( i );

    return true;
}

__global__ void sumOfSquares( int *num, int *result, clock_t * time )
{
    const int tid = threadIdx.x;
}

int cu1_init()
{
    std::cout << "Hello, World!" << std::endl;

    if( !InitCUDA() )
        return -1;
    printf( "InitCUDA is over.\n");

    return 0;
}
