#include "hip/hip_runtime.h"
//
// Created by hk001 on 2021/11/5.
//

#include "../../include/0_CU/cu2_addSum.cuh"



void generateNumbers_2( int *number, int size )
{
    for( int i = 0; i < size; i++ )
        number[i] = rand() % 10;
}

bool initCUDA_2()
{
    int count;

    hipGetDeviceCount(&count);

    if (count == 0) return false;

    int i;
    for (i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        if( hipGetDeviceProperties( &prop, i ) ==  hipSuccess )
            if( prop.major >= 1 )
                break;
    }

    if( i == count ) return false;

    hipSetDevice( i );

    return true;
}

__global__ static void sumOfSquares_2( int *num, int *result )
{
    int sum = 0;
    int i;
    for( i = 0; i < DATA_SIZE; i++ )
        sum += num[i] * num[i] * num[i] ;

    *result = sum;
}

int cu2_addSum()
{
    if( !initCUDA_2() ) return 0;

    generateNumbers_2( data, DATA_SIZE );

    int *gpuData, *result;

    hipMalloc( (void**)&gpuData, sizeof(int) * DATA_SIZE );
    hipMalloc( (void**)&result, sizeof(int) );
    hipMemcpy( gpuData, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice );

    sumOfSquares_2<<<1, 1, 0>>>( gpuData, result );

    int sum;
    hipMemcpy( &sum, result, sizeof( int ), hipMemcpyDeviceToHost );

    hipFree( gpuData );
    hipFree( result );
    printf( "GPUsum: %d \n", sum );

    sum = 0;
    for( int i = 0; i < DATA_SIZE; i++ )
        sum += data[i] * data[i] * data[i];
    printf( "CPUsum: %d \n", sum );

    return 0;
}


















