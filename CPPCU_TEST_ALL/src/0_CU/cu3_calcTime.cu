#include "hip/hip_runtime.h"
//
// Created by hk001 on 2021/11/5.
//

#include "../../include/0_CU/cu3_calcTime.cuh"


void generateNumbers_3( int *number, int size )
{
    for( int i = 0; i < size; i++ )
        number[i] = rand() % 10;
}

bool initCUDA_3()
{
    int count;

    hipGetDeviceCount(&count);

    if (count == 0) return false;

    int i;
    for (i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        if( hipGetDeviceProperties( &prop, i ) ==  hipSuccess )
            if( prop.major >= 1 )
                break;
    }

    if( i == count ) return false;

    hipSetDevice( i );

    return true;
}

__global__ static void sumOfSquares_3( int *num, int *result, clock_t *time )
{
    int sum = 0;
    int i;

    clock_t start = clock();

    for( i = 0; i < DATA_SIZE; i++ )
        sum += num[i] * num[i] * num[i] ;

    *result = sum;

    *time = clock() - start;
}

int cu3_calcTime()
{
    if( !initCUDA_3() ) return 0;

    generateNumbers_3( data, DATA_SIZE );

    int *gpuData, *result;
    clock_t *time;

    hipMalloc( (void**)&gpuData, sizeof(int) * DATA_SIZE );
    hipMalloc( (void**)&result, sizeof(int) );
    hipMalloc( (void**)&time, sizeof( clock_t ) );
    hipMemcpy( gpuData, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice );

    sumOfSquares_3<<<1, 1, 0>>>(gpuData, result, time);

    int sum;
    clock_t time_used;
    hipMemcpy( &sum, result, sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( &time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost );

    hipFree( gpuData );
    hipFree( result );
    hipFree( time );
    printf( "GPUsum: %d time: %ld \n", sum, time_used );

    sum = 0;
    for( int i = 0; i < DATA_SIZE; i++ )
        sum += data[i] * data[i] * data[i];
    printf( "CPUsum: %d \n", sum );

    return 0;
}























